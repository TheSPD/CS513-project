#include "hip/hip_runtime.h"
/*
* multNoShare.c
*
*/
#include "matrix.cuh"
#include <stdio.h>


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
// Parallel multiplication of Matrices

__global__ void ChainMatMulKernel(Matrix* Chain, int* Muls, Matrix* IntRes) {
	int threadId = blockIdx.x;
	int row = blockIdx.y;
	int col = blockIdx.z;
	int Cvalue = 0;

	int mulNum = Muls[threadId];

	if(row >= Chain[mulNum].height || col >= Chain[mulNum + 1].width) return;
	for (int e = 0; e < Chain[mulNum].width; ++e){
		Cvalue += ((Chain[mulNum].elements[row * Chain[mulNum].width + e]) * (Chain[mulNum + 1].elements[e * Chain[mulNum + 1].width + col]));
		Cvalue = Cvalue	% 256;
	}
	IntRes[threadId].elements[row * IntRes[threadId].width + col] = Cvalue;	   
}

void SequentialSelectionSortDouble(int* array, int* arrayOrder, int n) {
	int position, swap, swapOrder, d, c;

	for(c = 0 ; c < ( n - 1 ) ; c++) {
		position = c;
 
		for ( d = c + 1 ; d < n ; d++ ) {
			if ( array[position] < array[d] )
				position = d;
		}
		if(position != c) {
			swap = array[c];
			swapOrder = arrayOrder[c];
		 
			array[c] = array[position];
			arrayOrder[c] = arrayOrder[position];

			array[position] = swap;
			arrayOrder[position] = swapOrder;
		}
	}
}

void SequentialSelectionSort(int* array, int n) {
	int position, swap, d, c;

	for(c = 0 ; c < ( n - 1 ) ; c++) {
		position = c;
 
		for ( d = c + 1 ; d < n ; d++ ) {
			if ( array[position] > array[d] )
				position = d;
		}
		if(position != c) {
			swap = array[c];
		 
			array[c] = array[position];

			array[position] = swap;
		}
	}
}

Matrix ChainMatMul(Matrix* Chain, int numMats) {

	int n = numMats;
	Matrix Result;
	Matrix* h_Chain; // Only elements on device
	Matrix* d_Chain; // Array fully on device
	Matrix* h_IntRes; // Only elements on device
	Matrix* d_IntRes; // Array fully on device
	int* ChainDims;
	int* ChainDimOrder;
	int numDims;
	int* h_muls; // Array on host
	int* d_muls; // Array on device
	int numMuls = 0;
	
	h_Chain = (Matrix*)malloc(n*sizeof(Matrix));
	
	size_t size;
	hipError_t err;
	
	// Transfer from Chain to h_Chain
	for(int i = 0; i < n;++i) {	        
		h_Chain[i].width = Chain[i].width;
		h_Chain[i].height = Chain[i].height;
		size = h_Chain[i].width * h_Chain[i].height * sizeof(int);
		err = hipMalloc(&h_Chain[i].elements, size);
		//printf("CUDA malloc Chain[%d].elements: %s\n", i, hipGetErrorString(err));
		err = hipMemcpy(h_Chain[i].elements, Chain[i].elements, size, hipMemcpyHostToDevice);
		//printf("Copy Chain[%d].elements to device: %s\n", i, hipGetErrorString(err));
	}
		
	// Trasfer from h_Chain to d_Chain
	size = n * sizeof(Matrix);
	err = hipMalloc(&d_Chain, size);
	//printf("CUDA malloc Chain: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_Chain, h_Chain, size, hipMemcpyHostToDevice);
	

	while (n > 1) {
		// ************************** Find optimal multiplications ******************
		// Fill up ChainDims
		numDims = n - 1;
		numMuls = 0;
		ChainDims = (int*)malloc(numDims * sizeof(int));
		ChainDimOrder = (int*)malloc(numDims * sizeof(int));
		h_muls = (int*)malloc(numDims * sizeof(int));
		for(int i = 0; i < numDims; ++i) {
			ChainDims[i] = h_Chain[i].width;
			ChainDimOrder[i] = i;
		}

		// Sort ChainDims
		SequentialSelectionSortDouble(ChainDims, ChainDimOrder, numDims);

		// Select muls
		for(int i = 0, j = 0;i < numDims; ++i) {
			if(ChainDims[i] != 0 && (numMuls < 1024)) {
				h_muls[j] = ChainDimOrder[i];
				numMuls++;
				j++;
				for(int k = 0; k < numDims; k++){
					if(ChainDimOrder[k] == (ChainDimOrder[i] + 1) || ChainDimOrder[k] == (ChainDimOrder[i] - 1)) {
						ChainDims[k] = 0;
					}
				}
			}
		}
		free(ChainDims);
		free(ChainDimOrder);

		SequentialSelectionSort(h_muls, numMuls);
		printf("\nMultiplication choices : ");
		for(int i = 0; i < numMuls; ++i) {
			printf("Mat%d x Mat%d\t", h_muls[i], (h_muls[i]+1));
		}
		printf("\n");
		// **************************************************************************

		// ********************** Transfer stuff to Device **************************
		// Transfer muls on device
		err = hipMalloc(&d_muls, numMuls * sizeof(int));
		//printf("CUDA malloc Muls: %s\n", hipGetErrorString(err));
		err = hipMemcpy(d_muls, h_muls, numMuls * sizeof(int), hipMemcpyHostToDevice);
		//printf("Copy Muls to device: %s\n", hipGetErrorString(err));

		// Hold intermediate results on host with elements on device
		h_IntRes = (Matrix*)malloc(numMuls * sizeof(Matrix));
		
		// Allocate memory on device for the elements of h_IntRes
		for(int i = 0; i < numMuls; ++i) {
			h_IntRes[i].height = h_Chain[h_muls[i]].height;
			h_IntRes[i].width = h_Chain[h_muls[i] + 1].width;
			size_t size = h_IntRes[i].width * h_IntRes[i].height * sizeof(int);
			err = hipMalloc(&h_IntRes[i].elements, size);
			//printf("CUDA malloc IntRes[%d]: %s\n", i, hipGetErrorString(err));
		}
		
		// IntRes Fully on device
		size = numMuls * sizeof(Matrix);
		err = hipMalloc(&d_IntRes, size);
		//printf("CUDA malloc Chain: %s\n", hipGetErrorString(err));
		err = hipMemcpy(d_IntRes, h_IntRes, size, hipMemcpyHostToDevice);
		//printf("Copy Chain to device: %s\n", hipGetErrorString(err));
		
		// **************************************************************************

		// *************************** Actual Multiplication ************************
		dim3 dimGrid(numMuls, 256, 256);

		// Call to the kernel
		ChainMatMulKernel<<<dimGrid, 1>>>(d_Chain, d_muls, d_IntRes); 
		err = hipDeviceSynchronize();
		//printf("Run kernel: %s\n", hipGetErrorString(err));

		// **************************************************************************

		// ************************** Readying for next cycle ***********************
		// Update chain 
		for(int i = 0; i < numMuls;++i) {
			// Free device memory
			hipFree(h_Chain[h_muls[i]].elements);
			hipFree(h_Chain[h_muls[i] + 1].elements);
			
			// Update the chain
			h_Chain[h_muls[i]].height = h_IntRes[i].height;
			h_Chain[h_muls[i]].width = h_IntRes[i].width;
			h_Chain[h_muls[i]].elements = h_IntRes[i].elements;
		}
		
		// Reduce the size of the h_Chain array
		for(int i = 0; i < numMuls; ++i){
			h_Chain[h_muls[i]+1].width = 0;	
			h_Chain[h_muls[i]+1].height = 0;
		}

		for(int i = 0, j =0; i < n; ++i) {
			if(h_Chain[i+j].width == 0) {
				j++;
				n--;
			}
			h_Chain[i].width = h_Chain[i + j].width;	
			h_Chain[i].height = h_Chain[i + j].height;
			h_Chain[i].elements = h_Chain[i + j].elements;
		}
		// Small memory leak here - (but removing this is difficult)

		// Refresh d_Chain
		hipFree(d_Chain);
		
		size = n * sizeof(Matrix);
		err = hipMalloc(&d_Chain, size);
		//printf("CUDA malloc Chain: %s\n", hipGetErrorString(err));
		err = hipMemcpy(d_Chain, h_Chain, size, hipMemcpyHostToDevice);
		//printf("Copy Chain to device: %s\n", hipGetErrorString(err));
			
		// Free stuff
		free(h_muls);
		hipFree(d_muls);
		free(h_IntRes);
		hipFree(d_IntRes);

		// **************************************************************************
	}
	
	// Read Result from device memory
	Result.width = h_Chain[0].width;	
	Result.height = h_Chain[0].height;
	size = Result.width * Result.height * sizeof(int);
	Result.elements =  (int*)malloc(size);
	err = hipMemcpy(Result.elements, h_Chain[0].elements, size, hipMemcpyDeviceToHost);
	//printf("Copy Result off of device: %s\n",hipGetErrorString(err));
	
	hipFree(h_Chain[0].elements);
	hipFree(d_Chain);
	free(h_Chain);
	
	return Result;
}

// Usage: multNoShare a1 a2 b2
int main(int argc, char* argv[]){
	Matrix* Chain;
	Matrix Result;
	int* dims; 
	
	if(argc != 2) {
		printf("Please input in the following format\n multNoShare.out [#FileName] \n");
		return 0;
	}
		
	char const* const fileName = argv[1]; /* should check that argc > 1 */

	FILE* file = fopen(fileName, "r"); /* should check the result */
	char nLine[10];
	char line[256 * 5];
	int lineNums[256];

	// Read values from file
	int n = atoi(fgets(nLine, sizeof(nLine), file)) - 1;
	int sizeDim = sizeof(char) * (n+1) * 5;

	char* dimLine = (char *)malloc(sizeDim);
	fgets(dimLine, sizeof(char) * (n+1) * 5, file);
	char oneNum[5];
	dims = (int *) malloc((n+1)*sizeof(int));

	for(int i = 0, k = 0; dimLine[i] != '\0' ;++i) {
		int j = 0;
		for(; dimLine[i] != ' ' && dimLine[i]!= '\0'; ++j,++i){
			oneNum[j] = dimLine[i];
		}
		oneNum[j] = '\0';
		dims[k++] = atoi(oneNum);
	}

	Chain = (Matrix *) malloc(n*sizeof(Matrix));
	for(int i = 0; i < n; ++i) {
		Chain[i].height = dims[i];
		Chain[i].width = dims[i+1];
		Chain[i].elements = (int*)malloc(Chain[i].width * Chain[i].height * sizeof(int));
	}

	for(int k = 0; k < n; ++k)
		for(int i = 0; i < Chain[k].height; i++){
			fgets(line, sizeof(line), file);
			for(int p = 0, q = 0; line[p] != '\0' ;++p) {
				int r = 0;
				for(; line[p] != ' ' && line[p]!= '\0'; ++r,++p){
					oneNum[r] = line[p];
				}
				oneNum[r] = '\0';
				lineNums[q++] = atoi(oneNum);
			}	
			for(int j = 0; j < Chain[k].width; j++) {
				Chain[k].elements[i*Chain[k].width + j] = lineNums[j];
			}
		}


	fclose(file); 

	printf("Print up to a 10x10 portion of the matrices - to avoid clutter");
	for(int k = 0; k < n; ++k) {
		printf("\n Chain[%d] : %d x %d\n", k, Chain[k].height, Chain[k].width);
		for(int i = 0; i < min(10, Chain[k].height); i++){
			for(int j = 0; j < min(10, Chain[k].width); j++)
				printf("%d ", Chain[k].elements[i*Chain[k].width + j]);
			printf("\n");
		}
	}
	printf("\n");

	Result = ChainMatMul(Chain, n);
	
	// Print up to a 10x10 portion of the Result
	printf("\n Result : %d x %d\n", Result.height, Result.width);
	for(int i = 0; i < min(10, Result.height); i++){
		for(int j = 0; j < min(10, Result.width); j++)
			printf("%d ", Result.elements[i*Result.width + j]);
		printf("\n");
	}
	
}
